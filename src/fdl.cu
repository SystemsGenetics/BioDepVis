#include "hip/hip_runtime.h"
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <iostream>
#include "fdl.h"



const float MAX_DISPLACEMENT_SQR = 10.0f;



#define ELEM(data, cols, i, j) (data)[(i) * (cols) + (j)]



#define CUDA_SAFE_CALL(x)                              \
{                                                      \
	hipError_t result = x;                            \
	if ( result != hipSuccess )                       \
	{                                                  \
		std::cerr                                      \
			<< "CUDA Error: " #x " failed with error " \
			<< hipGetErrorString(result) << '\n';     \
		exit(1);                                       \
	}                                                  \
}



void * gpu_malloc(int size)
{
	void *ptr = nullptr;

	CUDA_SAFE_CALL(hipMalloc(&ptr, size));

	return ptr;
}



void gpu_free(void *ptr)
{
	CUDA_SAFE_CALL(hipFree(ptr));
}



void gpu_read(void *dst, void *src, int size)
{
	CUDA_SAFE_CALL(hipMemcpy(dst, src, size, hipMemcpyDeviceToHost));
}



void gpu_write(void *dst, void *src, int size)
{
	CUDA_SAFE_CALL(hipMemcpyAsync(dst, src, size, hipMemcpyHostToDevice));
}



__global__
void fdl_kernel_2d(int n, vec3_t *positions, vec3_t *positions_d, const bool *edge_matrix)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if ( i >= n )
	{
		return;
	}

	const float K_r = 0.2f;
	const float K_s = 1.0f;
	const float L = 2.2f;

	for ( int j = 0; j < n; j++ )
	{
		if ( i == j )
		{
			continue;
		}

		float dx = positions[j].x - positions[i].x;
		float dy = positions[j].y - positions[i].y;
		float dist = sqrtf(dx * dx + dy * dy);

		if ( dist != 0 )
		{
			float force = ELEM(edge_matrix, n, i, j)
				? K_s * (L - dist) / dist
				: K_r / (dist * dist * dist);

			positions_d[i].x -= force * dx;
			positions_d[i].y -= force * dy;

			positions_d[j].x += force * dx;
			positions_d[j].y += force * dy;
		}
	}
	__syncthreads();

	float dx = positions_d[i].x;
	float dy = positions_d[i].y;
	float disp_sqr = dx * dx + dy * dy;

	if ( disp_sqr > MAX_DISPLACEMENT_SQR )
	{
		dx *= sqrtf(MAX_DISPLACEMENT_SQR / disp_sqr);
		dy *= sqrtf(MAX_DISPLACEMENT_SQR / disp_sqr);
	}

	positions[i].x += dx;
	positions[i].y += dy;
	positions_d[i].x *= 0.1f;
	positions_d[i].y *= 0.1f;
}



__global__
void fdl_kernel_3d(int n, vec3_t *positions, vec3_t *positions_d, const bool *edge_matrix)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if ( i >= n )
	{
		return;
	}

	const float K_r = 0.2f;
	const float K_s = 1.0f;
	const float L = 2.2f;

	for ( int j = 0; j < n; j++ )
	{
		if ( i == j )
		{
			continue;
		}

		float dx = positions[j].x - positions[i].x;
		float dy = positions[j].y - positions[i].y;
		float dz = positions[j].z - positions[i].z;
		float dist = sqrtf(dx * dx + dy * dy + dz * dz);

		if ( dist != 0 )
		{
			float force = ELEM(edge_matrix, n, i, j)
				? K_s * (L - dist) / dist
				: K_r / (dist * dist * dist);

			positions_d[i].x -= force * dx;
			positions_d[i].y -= force * dy;
			positions_d[i].z -= force * dz;

			positions_d[j].x += force * dx;
			positions_d[j].y += force * dy;
			positions_d[j].z += force * dz;
		}
	}
	__syncthreads();

	float dx = positions_d[i].x;
	float dy = positions_d[i].y;
	float dz = positions_d[i].z;
	float disp_sqr = dx * dx + dy * dy + dz * dz;

	if ( disp_sqr > MAX_DISPLACEMENT_SQR )
	{
		dx *= sqrtf(MAX_DISPLACEMENT_SQR / disp_sqr);
		dy *= sqrtf(MAX_DISPLACEMENT_SQR / disp_sqr);
		dz *= sqrtf(MAX_DISPLACEMENT_SQR / disp_sqr);
	}

	positions[i].x += dx;
	positions[i].y += dy;
	positions[i].z += dz;
	positions_d[i].x *= 0.1f;
	positions_d[i].y *= 0.1f;
	positions_d[i].z *= 0.1f;
}



void fdl_2d_gpu(int n, vec3_t *positions, vec3_t *positions_d, const bool *edge_matrix)
{
	const int BLOCK_SIZE = 256;
	const int GRID_SIZE = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;

	fdl_kernel_2d<<<GRID_SIZE, BLOCK_SIZE>>>(
		n,
		positions,
		positions_d,
		edge_matrix);
	CUDA_SAFE_CALL(hipGetLastError());
}



void fdl_3d_gpu(int n, vec3_t *positions, vec3_t *positions_d, const bool *edge_matrix)
{
	const int BLOCK_SIZE = 256;
	const int GRID_SIZE = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;

	fdl_kernel_3d<<<GRID_SIZE, BLOCK_SIZE>>>(
		n,
		positions,
		positions_d,
		edge_matrix);
	CUDA_SAFE_CALL(hipGetLastError());
}

#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include "fdl.h"

void checkError(hipError_t err)
{
	if ( err != hipSuccess ) {
		fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(err));
		exit(1);
	}
}

void * gpu_malloc(int size)
{
	void *ptr = nullptr;

	hipMalloc(&ptr, size);
	checkError(hipGetLastError());

	return ptr;
}

void gpu_free(void *ptr)
{
	hipFree(ptr);
	checkError(hipGetLastError());
}

void gpu_read(void *dst, void *src, int size)
{
	hipMemcpy(dst, src, size, hipMemcpyDeviceToHost);
	checkError(hipGetLastError());
}

void gpu_write(void *dst, void *src, int size)
{
	hipMemcpy(dst, src, size, hipMemcpyHostToDevice);
	checkError(hipGetLastError());
}

void fdl_2d_gpu(int n, vec3_t *coords, vec3_t *coords_d, int *edge_matrix)
{
}

void fdl_3d_gpu(int n, vec3_t *coords, vec3_t *coords_d, int *edge_matrix)
{
}
